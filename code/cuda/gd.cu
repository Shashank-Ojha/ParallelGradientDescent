#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <hiprand/hiprand_kernel.h>

#include "regression.h"

void printCudaInfo()
{
    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}

__device__ __inline__ float
evaluateCuda(estimate_t* estimate, float x){
  return (estimate->b1)*x;
}

__device__ __inline__ float
getdB1Cuda(float x, float y, estimate_t* estimate){
  float prediction = evaluateCuda(estimate, x);
  return -2.0 * (y-prediction)*x;
}


__global__ void
setup_kernel(hiprandState *states) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    /* Each thread gets same seed, a different sequence
       number, no offset */
    hiprand_init(418, index, 0, &states[index]);
}

// Assumes the number of indexes is equal to N
__global__ void
sgd_step(int N, float* device_X, float* device_Y, estimate_t* device_estimates, hiprandState* states) {

  int index = blockIdx.x * blockDim.x + threadIdx.x;

  hiprandState localState = states[index];

  int pi = hiprand(&localState) % N;
  states[index] = localState;

  float db1 = (1.0 / static_cast<float>(N)) * getdB1Cuda(device_X[pi], device_Y[pi],
                                                    device_estimates + index);

  device_estimates[index].b1 -= (STEP_SIZE_STOCH * db1);
}

// Running SGD with all threads each sampling one point and averaging result
// after each SGD step. Checking convergence after each step
estimate_t* sgdCuda(int N, float* x, float* y, float alpha, float opt,
                    int blocks, int threadsPerBlock){

  float* device_X;
  float* device_Y;
  estimate_t* device_estimates;

  int totalThreads = blocks * threadsPerBlock;

  hiprandState *states;
  hipMalloc((void**)&states, totalThreads * sizeof(hiprandState));

  hipMalloc((void **)&device_X, sizeof(float) * N);
  hipMalloc((void **)&device_Y, sizeof(float) * N);
  hipMalloc((void **)&device_estimates, sizeof(estimate_t) * totalThreads);

  //might have to check this
  estimate_t* estimates = (estimate_t*)calloc(totalThreads, sizeof(estimate_t));

  hipMemcpy(device_X, x, N * sizeof(float),
             hipMemcpyHostToDevice);

  hipMemcpy(device_Y, y, N * sizeof(float),
             hipMemcpyHostToDevice);

  hipMemcpy(device_estimates, estimates, totalThreads * sizeof(estimate_t),
             hipMemcpyHostToDevice);

  setup_kernel<<<blocks, threadsPerBlock>>>(states);

  estimate_t* ret = (estimate_t*)malloc(sizeof(estimate_t));

  float upper = opt + (alpha/2.0)*opt;
  float lower = opt - (alpha/2.0)*opt;

  int num_steps = 0;
  while(true)
  {
    sgd_step<<<blocks, threadsPerBlock>>>(N, device_X, device_Y, device_estimates, states);

    hipMemcpy(estimates, device_estimates, totalThreads * sizeof(estimate_t),
               hipMemcpyDeviceToHost);

    ret -> b1 = 0.0;
    for(int j = 0; j < totalThreads; j++) {
      ret -> b1 += estimates[j].b1;
    }

    ret -> b1 /= static_cast<float>(totalThreads);

    if(num_steps > ITER_LIMIT || (lower < (ret -> b1) && (ret -> b1) < upper))
      break;

    num_steps += 1;
  }

  printf("Num iterations: %d\n", num_steps);

  return ret;
}
